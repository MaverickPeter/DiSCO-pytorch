#include <stdio.h>
#include <ctime>
#include <cassert>
#include <cmath>
#include <utility>
#include <vector>
#include <algorithm> 
#include <cstdlib>
#include <memory>
#include <iostream>
#include "hip/hip_runtime.h"


float __device__ rad2deg(float radians)
{
    return radians * 180.0 / M_PI;
}

float __device__ deg2rad(float degrees)
{
    return degrees * M_PI / 180.0;
}

float __device__ xy2theta(const float & _x, const float & _y )
{
    if ( _x >= 0 & _y >= 0) 
        return (180/M_PI) * atan(_y / _x);

    else if ( _x < 0 & _y >= 0) 
        return 180 - ( (180/M_PI) * atan(_y / (-_x)) );

    else if ( _x < 0 & _y < 0) 
        return 180 + ( (180/M_PI) * atan(_y / _x) );

    else if ( _x >= 0 & _y < 0)
        return 360 - ( (180/M_PI) * atan((-_y) / _x) );
} // xy2theta


// convert pointcloud to grid map
void __global__ point2gridmap(float* point, int* ring, int* sector, int* height, int d_size, int max_length, int max_height, int num_ring, int num_sector, int num_height) 
{
    int gid = threadIdx.x + blockDim.x*blockIdx.x;
    
    if(gid >= d_size) return;

    float gap_ring, gap_sector, gap_height;

    gap_ring = (float)max_length / (float)num_ring;
    gap_sector = 360.0/(float)num_sector;
    gap_height = 2.0 * (float)max_height / (float)num_height;

    float x, y, z;
    x = point[gid];
    y = point[gid + d_size];
    z = point[gid + 2 * d_size];
    
    if(x == 0.0)
        x = 0.0001;
    if(y == 0.0)
        y = 0.0001;
    if(z == 0.0)
        z = 0.0001;

    float theta = xy2theta(x, y);
    float faraway = sqrt(pow(x,2) + pow(y,2));

    int idx_ring = floor(faraway / gap_ring);
    int idx_sector = floor(theta / gap_sector);
    int idx_height = floor((z + (float)max_height) / gap_height);

    if(idx_ring >= num_ring)
       idx_ring = num_ring - 1;

    height[gid] = idx_height;
    ring[gid] = idx_ring;
    sector[gid] = idx_sector;

    __syncthreads();  
}

